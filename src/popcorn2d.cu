#include "hip/hip_runtime.h"

#include "cuda_globals.hpp"

#include "ppm.hpp"
#include "CsvWriter.h"

#include <sstream>
#include <iostream>
#include <chrono>
#include <time.h>
#include <cinttypes>
#include <math.h>

#define VERBOSE 0

/*test setup
 * Total Number of performed tests: numberofTest*talphaCount*numberOfRescaling
 * Number of tests performed with set talpha: numberOfTests
 * Number of tests performed with set resolution: numberOfTest*talphaCount
 */

#define numberOfRescalings 0 //sets number of resolution rescalings
#define EnableSafedialog 0 /*controlls if safedialog is displayed or skipped
                             if skipped, picture wont be safed to file*/

static constexpr bool UseAtomics=false;
static constexpr bool RenderTrace=false;

template<typename T>
struct Parameters
{
  unsigned width = 256;
  unsigned height = 256;
  unsigned n;

  const T y1 = 5.0, y0 = -5.0, x0 = 5.0, x1 = -5.0; //sets zoom
  const T t0 = 31.1;
  const T t1 = -43.4;
  const T t2 = -43.3;
  const T t3 = 22.2;

  T talpha = 0.01;
};


//image settings
const uint32_t ITERATION = 64;
const uint32_t RES_EXPANSION = 128; //expands image resolution for each test
const uint32_t WIDTH_START  = 256;
const uint32_t HEIGHT_START = 256;

const double PI = 3.141592653589793;
static hipEvent_t custart, cuend;


using namespace std;

// from world-space to image-space
template<typename T>
__device__
unsigned unmap( T v, const T v0, const T v1, const T len)
{
  return static_cast<unsigned>( (v-v0)/(v1-v0)*len );
}

// from image-space to world-space
template<typename T>
__device__
T map( unsigned v, const T v0, const T v1, const T len)
{
  return static_cast<T>(v)/len*(v1-v0)+v0;
}


/// HSL [0:1] to RGB {0..255}, from http://stackoverflow.com/questions/4728581/hsl-image-adjustements-on-gpu
template<typename T>
__device__
void hsl2rgb( T* _data, unsigned n, float hue, float sat, float lum )
{
  const float onethird = 1.0 / 3.0;
  const float twothird = 2.0 / 3.0;
  const float rcpsixth = 6.0;

  float xtr = rcpsixth * (hue - twothird);
  float xtg = 0.0;
  float xtb = rcpsixth * (1.0 - hue);

  if (hue < twothird) {
    xtr = 0.0;
    xtg = rcpsixth * (twothird - hue);
    xtb = rcpsixth * (hue      - onethird);
  }

  if (hue < onethird) {
    xtr = rcpsixth * (onethird - hue);
    xtg = rcpsixth * hue;
    xtb = 0.0;
  }

  xtr = __saturatef(xtr);
  xtg = __saturatef(xtg);
  xtb = __saturatef(xtb);

  float sat2   =  2.0 * sat;
  float satinv =  1.0 - sat;
  float luminv =  1.0 - lum;
  float lum2m1 = (2.0 * lum) - 1.0;
  float ctr    = (sat2 * xtr) + satinv;
  float ctg    = (sat2 * xtg) + satinv;
  float ctb    = (sat2 * xtb) + satinv;

  if (lum >= 0.5) {
    _data[0] = ((luminv * ctr) + lum2m1);
    _data[n] = ((luminv * ctg) + lum2m1);
    _data[2*n] = ((luminv * ctb) + lum2m1);
  }else {
    _data[0] = (lum * ctr);
    _data[n] = (lum * ctg);
    _data[2*n] = (lum * ctb);
  }
}


template<typename T>
__global__
void d_colorImage(T* _data, const Parameters<T> _params)
{
  unsigned j;
  T* red   = _data;
  T* green = _data+_params.n;
  T* blue  = _data+2*_params.n;
  for (j = blockIdx.x * blockDim.x + threadIdx.x;
       j < _params.n;
       j += blockDim.x * gridDim.x)
  {
    if(RenderTrace) {
      // red[j]   = 1.0-red[j];
      // green[j] = 1.0-green[j];
      // blue[j]  = 1.0-blue[j];
    }
    else {
      T colors[3];
      T density = sqrt(_data[j]);
      colors[0] = pow(density,0.4);
      colors[1] = pow(density,1.0);
      colors[2] = pow(density,1.4);
      // check if color values in range of [0,1], else correct
      for(int count = 0; count <  3; ++count){
        if (colors[count] > 1 ){
          colors[count] = 1;
        }else if(colors[count] < 0){
          colors[count] = 0;
        }
      }

      red[j]   = 1.0-0.5*colors[0];
      green[j] = 1.0-0.2*colors[1];
      blue[j]  = 1.0-0.4*colors[2];
      /*unsigned char d = 255*data.buffer[j];
        ptr[j].x = d;
        ptr[j].y = d;
        ptr[j].z = d;*/
    }
  }
}

template<typename T>
__global__
void d_computeImage(T* _data, const Parameters<T> _params)
{
  unsigned i,j;

  for (i = blockIdx.y * blockDim.y + threadIdx.y;
       i < _params.height;
       i += blockDim.y * gridDim.y)
  {
    for (j = blockIdx.x * blockDim.x + threadIdx.x;
         j < _params.width;
         j += blockDim.x * gridDim.x)
    {

      T xk = map(j, _params.x0, _params.x1, (T)_params.width);
      T yk = map(i, _params.y0, _params.y1, (T)_params.height);
      // T xk = (T) j / _params.width * (_params.x1 - _params.x0) + _params.x0;
      // T yk = (T) i / _params.height * (_params.y1 - _params.y0) + _params.y0;


      for(unsigned t=0; t<ITERATION; ++t) {
        xk += _params.talpha * (cos( _params.t0 * _params.talpha + yk + cos(_params.t1 * _params.talpha + PI * xk)));
        yk += _params.talpha * (cos( _params.t2 * _params.talpha + xk + cos(_params.t3 * _params.talpha + PI * yk)));

        int px = unmap(xk, _params.x0, _params.x1, (T)_params.width);
        int py = unmap(yk, _params.y0, _params.y1, (T)_params.height);
        if (px>=0 && py>=0 && px<_params.width && py<_params.height) {
          unsigned offset = px+py*_params.width;
          T v = 0.001;
          if( RenderTrace ) {
            if( (i&31)==0 && (j&31)==0 ) {
              hsl2rgb(_data+offset, _params.n, 0.5*(float(t)/63)+0.25, 1.0, 0.6);
            }

          }
          else {

            if(UseAtomics) {
              atomicAdd(_data+offset, v); // just density
            } else {
              _data[offset] += v;
            }
          }
        }
      } // for
    }
  }
}


template<typename T>
__global__
void d_computeImage_1D(T* _data, const Parameters<T> _params)
{
  unsigned i;
  for (i = blockIdx.x * blockDim.x + threadIdx.x;
       i < _params.n;
       i += blockDim.x * gridDim.x)
  {
    T xk = map(i%_params.width, _params.x0, _params.x1, (T)_params.width);
    T yk = map(i/_params.height, _params.y0, _params.y1, (T)_params.height);
    // T xk = (T) j / _params.width * (_params.x1 - _params.x0) + _params.x0;
    // T yk = (T) i / _params.height * (_params.y1 - _params.y0) + _params.y0;

    for(unsigned t=0; t<ITERATION; ++t) {
      xk += _params.talpha * (cos( _params.t0 * _params.talpha + yk + cos(_params.t1 * _params.talpha + PI * xk)));
      yk += _params.talpha * (cos( _params.t2 * _params.talpha + xk + cos(_params.t3 * _params.talpha + PI * yk)));

      int px = unmap(xk, _params.x0, _params.x1, (T)_params.width);
      int py = unmap(yk, _params.y0, _params.y1, (T)_params.height);
      if (px>=0 && py>=0 && px<_params.width && py<_params.height) {
        unsigned offset = px+py*_params.width;
        T v = 0.001;
        if( RenderTrace ) {
          if( (i&31)==0 ) {
            hsl2rgb(_data+offset, _params.n, 0.5*(float(t)/63)+0.25, 1.0, 0.6);
          }

        }
        else {

          if(UseAtomics) {
            atomicAdd(_data+offset, v); // just density
          } else {
            _data[offset] += v;
          }
        }
      }
    } // for
  }
}



template<typename T>
double launch_kernel(T* _data, const Parameters<T>& _params)
{
  int numSMs;
  int devId = 0;
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, devId);

  dim3 threads( 16, 16 );
  dim3 threads1d( 128 );
  dim3 blocks( 32*numSMs );
  size_t num_bytes;
  hipError_t err;

  CHECK_CUDA(hipEventRecord(custart));
  auto start_time = std::chrono::high_resolution_clock::now();

//  d_computeImage<<<blocks, threads>>>(_data, _params);
  d_computeImage_1D<<<blocks, threads1d>>>(_data, _params);

  CHECK_CUDA(hipEventRecord(cuend));
  CHECK_CUDA( hipEventSynchronize(cuend) );
  auto end_time = std::chrono::high_resolution_clock::now();

  d_colorImage<<<blocks, threads1d>>>(_data, _params);

  return std::chrono::duration<double, std::milli>(end_time - start_time).count();

  // float ms = 0.0f;
  // CHECK_CUDA( hipEventElapsedTime(&ms, custart, cuend) );
  // return ms;
}


/**
 *
 */
template<typename T>
void alloc_buffer(T** _data, unsigned n)
{
  if(_data && *_data) {
    CHECK_CUDA( hipFree(*_data) );
    CHECK_CUDA( hipEventDestroy(custart) );
    CHECK_CUDA( hipEventDestroy(cuend) );
  }
  CHECK_CUDA( hipMalloc(_data, 3*n*sizeof(T)) );
  CHECK_CUDA( hipEventCreate(&custart) );
  CHECK_CUDA( hipEventCreate(&cuend) );
}

/**
 *
 */
template<typename T>
void init_buffer(T* _data, unsigned n)
{
  CHECK_CUDA( hipMemset(_data, 0.0, 3*n*sizeof(T)));
  CHECK_CUDA( hipDeviceSetCacheConfig(hipFuncCachePreferL1) );
}
template<typename T>
void download(T* _image, T* _data, unsigned n) {
  CHECK_CUDA(hipMemcpy(_image, _data, 3*n*sizeof(T), hipMemcpyDeviceToHost));
}

/**
 *
 */
template<typename T>
void cleanup_cuda(T* _data)
{
  if(_data) {
    CHECK_CUDA( hipFree(_data) );
    CHECK_CUDA( hipEventDestroy(custart) );
    CHECK_CUDA( hipEventDestroy(cuend) );
  }
}

int main(int argc, char **argv) {
  char buffer[17];
  string fname;
  char in;
  int flag = 0;
  int numberOfTests = 20;
  float dispersion = 0;
  uint32_t w = WIDTH_START;
  uint32_t h = HEIGHT_START;
  // dispersion initilation
  float talphaStart = 0.0;   		// sets start value for dispersion
  float talphaIncrement = 0.1;		// 0.01		// sets values by wich talpha is incremented
  float talphaCount = 2;			// sets how often talpha is incremented
  Parameters<float> params;
  float* data = nullptr;

  //checking cmd-line for arguments and override settings if necessary
  if (argc >= 2)
	  fname = std::string(argv[1]);
  if (argc >= 3)
	  talphaStart = atof(argv[2]);
  if (argc >= 4)
	  talphaIncrement = atof(argv[3]);
  if (argc >= 5)
	  talphaCount = atoi(argv[4]);
  if (argc >= 6)
    w = atoi(argv[5]);
  if (argc >= 7)
    h = atoi(argv[6]);
  if (argc >= 8)
    numberOfTests = atoi(argv[7]);

  if(talphaCount<1)
    talphaCount = 1;

  //creates log <- size depending on numberOfTests
  float log[numberOfTests + 3];//Height,Width,talpha,Test1,Test2...,Testn
  uint32_t img_size = w*h;
  //Output file
  std::CsvWriter Output;
  //generating first line for CSV-File (headline)
  std::stringstream sstr;
  sstr << "Width, Height, talpha,";
  for (int count = 0; count < numberOfTests; ++count){
    sstr << "Test"<<count<<",";
  }

  Output.addLineString(sstr.str());
  //the image is computed numberOfRescaling*numberOfTest-Times
  for (int rescaleCount = 0; rescaleCount <= numberOfRescalings; ++rescaleCount) {
  	//rescaling image
    //float* image = new float[3*img_size];
  	//performing computation numberOfTests-times with rescaled resolution
  	if (numberOfTests < 1){
  		return 100;
  	}
  	//set height and with in log
  	log[0] = w;
  	log[1] = h;
    params.width = w;
    params.height = h;
    params.n = h*w;

    alloc_buffer(&data, params.n);

  	for(int count = 0; count < talphaCount; ++count) {
  		//increment talpha
      if ( count > -1){
        dispersion = talphaStart + count * talphaIncrement;
      }
      //write talpha to log
      log[2] = dispersion;
      params.talpha = dispersion;
      double duration = 0;

      //compute image numberOfTests-times + 1 warmup
      for(int testNumber = -1; testNumber < numberOfTests; ++testNumber){
        //initImage(image, w, h);
        init_buffer(data, params.n);
        {
          duration = launch_kernel(data, params);
        }
        if(testNumber<0) // warmup
          continue;
        log[testNumber + 3] = duration;
      }

      for(int testNumber = 0; testNumber < numberOfTests; ++testNumber)
      {
        cout <<"Test "<< testNumber+1 <<" executed in "<< log[testNumber+3] << " ms; Resolution "<<w<<"x"<<h<<" talpha "<<log[2]<< endl;
      }
      Output.addLineValues(log, numberOfTests + 3);
      dispersion = 0;
  	}

#if EnableSafedialog == 1
    if(rescaleCount == numberOfRescalings) {
      //savedialog for last picture
      // cout << "Save file? j/n" << endl;
      // std::cin >> in;
      // if ( in == 'j') {
      //   flag = 1;
      // }
      // if (flag == 1){
        float* image = new float[3*img_size];
        download(image, data, params.n);
        ImageWriter::PPM::writeRGB(image, w, h, fname + ".png");
        cout <<"Saved to "+ fname + ".png.\n";
        delete[] image;
//      }
    }
#endif
  	//delete[] image;
    cleanup_cuda(data);
    data = nullptr;
    w += RES_EXPANSION;
    h += RES_EXPANSION;
    img_size = w*h;
  }
  cout << endl;
  //Write Log to CSV-file

  if (fname.length() < 1)
	  Output.writeToCSV("result");
  else
	  Output.writeToCSV(fname);

  CHECK_CUDA(hipDeviceReset());

  //exit programm
  return 0;
}
